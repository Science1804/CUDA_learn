#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__global__ void gid_2d_block(int * input)
{

	int tid = threadIdx.x;
	int block_offset = blockIdx.x*blockDim.x;
	int row_offset = gridDim.x*blockDim.x*blockIdx.y;
	int gid = tid + row_offset + block_offset ;

	printf("blockIdx.x : %d, blockIdx.y : %d, threadIdx.x : %d, gid : %d, data : %d \n",
		blockIdx.x, blockIdx.y , tid, gid, input[gid]);

}


int main()
{

	int array_size=16;
	int array_byte_size = sizeof(int) * array_size ;
	int h_data[] = {23,9,4,53,65,12,1,33,22,43,56,4,76,81,94,32};

	int * d_data;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

	dim3 block(4);
	dim3 grid(2,2);

	gid_2d_block <<<grid,block>>> (d_data);
	hipDeviceSynchronize();

	hipDeviceReset();


}

