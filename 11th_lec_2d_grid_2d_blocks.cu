#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__global__ void grid2d_2d_block(int * input)

{

	int tid = threadIdx.x + blockDim.x*threadIdx.y;
	int block_offset = blockIdx.x*blockDim.x*blockDim.y;
	int row_offset = blockDim.x*blockDim.y*gridDim.x*blockIdx.y;
	int gid = tid + block_offset + row_offset;

	printf("blockIdx.x : %d, blockIdx.y : %d, threadIdx.x : %d, gid: %d, value : %d \n",
		blockIdx.x,blockIdx.y,threadIdx.x,gid,input[gid]);	


}


int main()
{

	int array_size = 16;
	int array_byte_size = sizeof(int)*array_size;
	int h_data[] = {10,20,30,40,5,6,7,8,9,10,11,12,13,14,15,16};

	int * d_data;
	hipMalloc((void**)&d_data,array_byte_size);
	hipMemcpy(d_data,h_data, array_byte_size, hipMemcpyHostToDevice);

	dim3 block(2,2);
	dim3 grid(2,2);

	grid2d_2d_block <<< grid,block >>> (d_data) ;
	hipDeviceSynchronize();

	hipDeviceReset();

}
