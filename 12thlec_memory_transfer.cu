#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#include <cstring>
#include <time.h>

__global__ void memTrnasfer(int * input)
{
	int gid = threadIdx.x + blockIdx.x*blockDim.x;

	printf("tid : %d, gid : %d, value : %d \n",
	threadIdx.x, gid , input[gid]);

}

int main()
{
	int array_size = 128;
	int byte_size = sizeof(int) * array_size;

	int * h_input;
	h_input = (int*)malloc(byte_size);

	time_t t;
	srand((unsigned)time(&t));

	for (int i=0; i < array_size;i++)
	{
		h_input[i] = (int)(rand() & 0xff);
	}

	int * d_input;
	hipMalloc((void**)&d_input,byte_size);
	
	hipMemcpy(d_input,h_input,byte_size,hipMemcpyHostToDevice);

	dim3 block(64);
	dim3 grid(2);

	memTrnasfer <<< grid,block>>> (d_input);

	hipDeviceSynchronize();

	hipFree(d_input);
	free(h_input);
	hipDeviceReset();
	return 0;
}

