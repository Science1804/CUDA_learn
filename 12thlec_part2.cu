#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <cstring>

__global__ void memTransfer( int * input, int size)
{

	int gid = threadIdx.x + blockDim.x*blockIdx.x;

	if (gid < size)
	{

		printf("tid : %d, gid : %d, value : %d, \n",
			threadIdx.x,gid,input[gid]);
 	}
	
}


int main()
{

	int array_size=150;
	int byte_size = sizeof(int)*array_size;

	int * h_data;
	h_data = (int *)malloc(byte_size);

	time_t t;
	srand((unsigned)time(&t));

	for(int i=0;i < array_size;i++)
	{
		h_data[i] = (int)(rand() & 0xff);
	}


	int * d_data;
	hipMalloc((void **)&d_data,byte_size);

	hipMemcpy(d_data,h_data,byte_size,hipMemcpyHostToDevice);

	dim3 block(32);
	dim3 grid(5);

	memTransfer <<< grid,block >>> (d_data , array_size) ;
	hipDeviceSynchronize();
	
	hipFree(d_data);
	free(h_data);
	hipDeviceReset();
	return 0;
}
