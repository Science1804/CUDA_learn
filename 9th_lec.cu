#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>


__global__ void global_uniq_id(int * input)
{

	int tid = threadIdx.x;
	int offset = blockIdx.x * blockDim.x ;
	int gid = tid + offset ;
	
	printf("blockIdx.x : %d, blockIdx.y : %d, threadIdx.x : %d, gid: %d, value : %d \n",
		blockIdx.x,blockIdx.y,threadIdx.x,gid,input[gid]);
}

int main()
{

	int array_size = 16;
	int array_byte_size = sizeof(int) * array_size;
	int h_data[] = {23,9,4,53,65,12,1,33,87,45,23,12,342,56,44,99};

	for ( int i = 0; i < array_size; i++)
	{
		printf("%d ", h_data[i]);
	
	}
	printf("\n \n");

	int * d_data;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

	dim3 block(4);
	dim3 grid(2,2);

	global_uniq_id <<< grid,block >>> (d_data);
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}

