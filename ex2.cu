#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>


__global__ void ex3_1D_block_1D_grid(int * input)
{
	int tid_x = threadIdx.x ;
	int block_offset= blockIdx.x*blockDim.x;
	
	int gid = tid_x + block_offset;
	printf(" Value : %d, gid : %d \n",
		input[gid],gid);

}

__global__ void ex3_2D_block_1D_grid(int * input)
{
	int tid = threadIdx.x + threadIdx.y*blockDim.x ;
	int block_offset = blockIdx.x*blockDim.x*blockDim.y ;
	// int row_offset = gridDim.x*blockIdx.y*blockDim.x*blockDim.y;

	int gid = tid + block_offset;

	printf(" Value : %d, gid : %d \n",
		input[gid],gid);
}

__global__ void ex3_3D_block_1D_grid( int * input)
{
	int tid = threadIdx.x + threadIdx.y*blockDim.x + \
				threadIdx.z*blockDim.x*blockDim.y;
	// int block_offset = blockIdx.x*blockDim.x*blockDim.y*blockDim.z ;
				
	int gid = tid ; 

	printf(" Value : %d, gid : %d \n",
		input[gid],gid);
}

__global__ void ex3_1D_block_2D_grid( int * input)
{
	int tid = threadIdx.x ;
	// int block_offset= blockIdx.x*blockDim.x;

	int row_offset = gridDim.x ;
				
	int gid = tid + row_offset ;

	printf(" Value : %d, gid : %d \n",
		input[gid],gid);
}


int main()
{

	int array_size = 64;
	int size = sizeof(int)*array_size;

	int * h_input;
	h_input = (int*)malloc(size);

	for (int i=0; i < array_size;i++)
	{
		h_input[i] = i;
	}

	int * d_input;
	hipMalloc((void**)&d_input,size);

	hipMemcpy(d_input,h_input,size,hipMemcpyHostToDevice);

	// dim3 grid1(1);
	// dim3 block1(64);
	// dim3 block2(16,4);
	// dim3 block3(4,4,4);

	dim3 grid2(2,1);
	dim3 block1(32);
	
	// ex3_1D_block_1D_grid <<< grid1,block1 >>> (d_input);

	// ex3_2D_block_1D_grid <<< grid1 , block2 >>> (d_input);

	// ex3_3D_block_1D_grid <<< grid1 , block3 >>> (d_input);

	ex3_1D_block_2D_grid <<< grid2 , block1 >>> (d_input);



	hipDeviceSynchronize();

	hipFree(d_input);
	free(h_input);
	hipDeviceReset();
	return 0;
}
