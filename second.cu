#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""


#include <stdio.h>


__global__ void second()
{
    printf("Hello CUda 2 \n");
}

int main()
{
    dim3 block(4);
    dim3 grid(8);

    second << <grid , block >> > ();

    hipDeviceSynchronize();

    hipDeviceReset();

}