#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void thread_array(int * input)
{
	int tid = threadIdx.x;
	printf(" threadIdx.x : %d, value : %d \n",tid,input[tid]);
}

int main()
{

	int array_size=8;
	int array_byte_size = sizeof(int) * array_size;
	int h_data[] = {23,9,4,53,65,12,1,33} ;

	for (int i= 0; i < array_size; i++)
	{
		printf("%d ", h_data[i]);
	}

	printf("\n \n");

	int * d_data ;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data,h_data,array_byte_size, hipMemcpyHostToDevice);

	dim3 block(8);
	dim3 grid(1);

	thread_array <<<grid,block>>> (d_data);
	hipDeviceSynchronize();

	hipDeviceReset();

}
