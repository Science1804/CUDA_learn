#include "hip/hip_runtime.h"



#include <stdio.h>



__global__ void multiple_blocks()
{
	printf(" Hey this is being executed in different blocks \n");
}

int main()
{
	int nx,ny;
	nx = 16;
	ny = 4 ;

	dim3 block(8,2,1);
	dim3 grid(nx/block.x , ny/block.y ,1);

	multiple_blocks <<<grid,block>>> ();

	hipDeviceSynchronize();

	hipDeviceReset();
}

